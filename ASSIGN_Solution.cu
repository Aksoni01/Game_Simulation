
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here
__global__ void dFiring(int *gpucoordX, int *gpucoordY, int *gpuScore, int *gpuHealth, int T, int M, int *health,int *tank_alive,long int round) {
   
    *tank_alive=T;

    if(round % T ==0 || health[blockIdx.x] <= 0 ) return;

    __shared__ long long int previous_distance;
    previous_distance=LLONG_MAX;
    __syncthreads();
    int source = blockIdx.x ,destination = (blockIdx.x + round) %T;

    long long int global_dest_id = (long long int)gpucoordY[destination] * M + gpucoordX[destination];
    long long int global_source_id =(long long int) gpucoordY[source] * M + gpucoordX[source];
    long long int global_temp_id =(long long int)  gpucoordY[threadIdx.x] * M + gpucoordX[threadIdx.x];
    long long int curr= abs(global_temp_id - global_source_id );

    long long int val = (long long int) gpucoordX[source] * (gpucoordY[destination] - gpucoordY[threadIdx.x]) +
                            (long long int)  gpucoordX[destination] * (gpucoordY[threadIdx.x] - gpucoordY[source]) +
                            (long long int)  gpucoordX[threadIdx.x] * (gpucoordY[source] - gpucoordY[destination]);

    if( (health[threadIdx.x]>0 && global_dest_id > global_source_id && global_temp_id > global_source_id &&  val == 0 ) || (health[threadIdx.x]>0 && global_dest_id < global_source_id && global_temp_id < global_source_id && val == 0 ) ){
                atomicMin(&previous_distance, curr );
    }
    __syncthreads();

    if(  (global_dest_id > global_source_id && global_temp_id > global_source_id && previous_distance == curr ) || (global_dest_id < global_source_id && global_temp_id < global_source_id &&  previous_distance == curr) ){
        atomicSub(&gpuHealth[threadIdx.x],1);
        atomicAdd(&gpuScore[source],1);
    }
}

__global__ void dCount(int * gpuHealth,int *tank_alive){
  if(gpuHealth[threadIdx.x]<=0){
    atomicSub(tank_alive,1);
  }
}


__global__ void memoryallote(int * gpuHealth,int * gpuScore,int H){
  gpuHealth[threadIdx.x]=H;
  gpuScore[threadIdx.x]=0;
}


//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
  
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************
    int *gpucoordX,*gpucoordY,*gpuScore,*gpuHealth;
    hipMalloc(&gpucoordX,T * sizeof(int));
    hipMalloc(&gpucoordY,T * sizeof(int));
    hipMalloc(&gpuScore,T * sizeof(int));
    hipMalloc(&gpuHealth,T * sizeof(int));
    hipMemcpy(gpucoordX,xcoord,T * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpucoordY,ycoord,T * sizeof(int),hipMemcpyHostToDevice);

    
    memoryallote<<<1,T>>>(gpuHealth,gpuScore,H);
    hipDeviceSynchronize();

    int *tank_alive;
    long int round=1;
    hipHostAlloc(&tank_alive, sizeof(int), 0); 
    int *health;
    hipMalloc(&health,T * sizeof(int));

     do{
      hipMemcpy(health, gpuHealth, T * sizeof(int), hipMemcpyDeviceToDevice);
      dFiring<<<T, T>>>(gpucoordX, gpucoordY, gpuScore, gpuHealth, T, M+1, health,tank_alive,round);
      dCount<<<1, T>>>(gpuHealth, tank_alive);
      hipDeviceSynchronize();
      round++;
    }while (*tank_alive >= 2 );

    hipMemcpy(score,gpuScore,T * sizeof(int),hipMemcpyDeviceToHost);

    hipFree(health);
    hipFree(gpuHealth);
    hipFree(gpucoordX);
    hipFree(gpucoordY);
    hipFree(gpuScore);
    hipHostFree(tank_alive);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}
